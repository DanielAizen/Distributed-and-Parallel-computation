#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__device__ int IsConservativeGPU(char c1, char c2);

__device__ char IsSemiConservativeGPU(char s);

__device__ int MyStrlen(const char *str);

__device__ int MyStrchr(const char *s, char c);

__global__ void CreateMutationGPU(char *d_Seq1, char *d_Mutant_Seq2, double *d_Weights, int h_Offset, int h_Sort_Order, int h_Size2)
{ 
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < h_Size2)
    {
        int flagCons = 0;
        char cSC = 0; //char semi conservative
        if (h_Sort_Order == 1)
        {
            flagCons = IsConservativeGPU(d_Seq1[i + h_Offset], d_Mutant_Seq2[i]);
            if (flagCons == 0)
            {
                d_Mutant_Seq2[i] = d_Seq1[i + h_Offset];
            }
        }
        else
        {
            flagCons = IsConservativeGPU(d_Seq1[i + h_Offset], d_Mutant_Seq2[i]);
            if (flagCons == 0)
            {
                if (d_Weights[2] > d_Weights[3])
                {
                    cSC = IsSemiConservativeGPU(d_Seq1[i + h_Offset]);
                    
                    if (cSC != '$')
                    {
                        d_Mutant_Seq2[i] = cSC;
                    }
                }
                else{
                    d_Mutant_Seq2[i] = 'Z';
                }
            }
        }
        
    }
}
__device__ int IsConservativeGPU(char c1, char c2)
{
    const char *conservative_Group[CONSERVATIVE] = {"NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF"};
    int cmp1, cmp2;
    for (int j = 0; j < CONSERVATIVE; j++)
    {
        cmp1 = MyStrchr(conservative_Group[j],c1);
        cmp2 = MyStrchr(conservative_Group[j],c2);
        if(cmp1 == 0 && cmp2 == 0){
            return 1;
        }
    }
    return 0;
}
__device__ char IsSemiConservativeGPU(char s)
{
    const char *semi_Conservative_Group[SEMI_CONSERVATIVE] = {"SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM"};
    int cmp;
    for (int j = 0; j < SEMI_CONSERVATIVE; j++)
    {
        cmp = MyStrchr(semi_Conservative_Group[j], s);
        if(cmp == 0){
            for(int k =0; k < MyStrlen(semi_Conservative_Group[j]); k++){
                if(semi_Conservative_Group[j][k] != s && IsConservativeGPU(s , semi_Conservative_Group[j][k]) == 0)
					return semi_Conservative_Group[j][k];
            }
        }
    }
    return '$';
}
__device__ int MyStrlen(const char *str)
{
    int len = 0;
    while (*(str++))
        (len)++;
    return len;
}
__device__ int MyStrchr(const char *s, char c) {
    while (*s != c) {
        if (!*s++) {
            return 1;
        }
    }
    return 0;
}

int computeOnGPU(char *h_Seq1, char *h_Mutant_Seq2, int h_Offset, int h_Sort_Order, double *h_Weights, int h_Len_Seq1, int h_Len_Seq2)
{
    hipError_t err = hipSuccess;
    size_t size1 = h_Len_Seq1 * sizeof(char);
    size_t size2 = h_Len_Seq2 * sizeof(char);

    char *d_Seq1;
    char *d_Mutant_Seq2;
    err = hipMalloc((void **)&d_Seq1, size1);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for seq1 - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_Mutant_Seq2, size2);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for seq2 - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    double *d_Weights;
    size_t sizeW = 4 * sizeof(double);
    err = hipMalloc((void **)&d_Weights, sizeW);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for seq2 - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Copy data from host to the GPU memory
    err = hipMemcpy(d_Seq1, h_Seq1, size1, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device for seq1 - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_Mutant_Seq2, h_Mutant_Seq2, size2, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device for seq2 - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_Weights, h_Weights, sizeW, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device for weights - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   
    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (h_Len_Seq2 + threadsPerBlock - 1) / threadsPerBlock;
    CreateMutationGPU<<<blocksPerGrid, threadsPerBlock>>>(d_Seq1, d_Mutant_Seq2, d_Weights, h_Offset, h_Sort_Order, h_Len_Seq2);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch CreateMutationGPU kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
    // Copy the result from GPU to the host memory.
    err = hipMemcpy(h_Mutant_Seq2, d_Mutant_Seq2, size2, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free allocated memory on GPU
    if (hipFree(d_Seq1) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(d_Mutant_Seq2) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(d_Weights) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}
